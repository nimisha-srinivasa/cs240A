/* Compute the SVD of a matrix */

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <string.h>
// #define ROWS 31568
// #define COLS 51
#define ROWS 6
#define COLS 3
#define FILENAME "data.txt"
#define MIN(x, y) (((x) < (y)) ? (x) : (y))
#define N_ITER 1

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

void fill(float *p, int n) {
    // This will be replaced by retrieving the data...
    for (int i = 0; i < n; i++) {
        p[i] = (float) (2.0*drand48() + 1.0);
    }
}

void print_matrix(int m, int n, float *A, int lda, const char *name) {
    printf("================A===============================\n");
    for(int row = 0; row < m; row++) {
        for(int col = 0; col < n; col++) {
            float Areg = A[row + col*lda];
            printf("%f\t", Areg);
        }
        printf("\n");
    }
    printf("================end of A===============================\n");
}

void readMatrixFromFile(float *p, int lda){
    FILE *myFile;
    char *filename=FILENAME;
    myFile = fopen(filename, "r");
    if (myFile == NULL)
    {
        printf("Error Reading File\n");
        exit (0);
    }

    char *line=NULL;
    char *word=NULL;
    float attr;
    size_t len = 0;
    ssize_t read;
    int row,col;

    //fill the matrix
    row=0;
    while (((read = getline(&line, &len, myFile)) != -1) && row<ROWS) {
        col=0;
        do{
            word=strsep(&line,",");
            attr = atof(word);
            p[row + col*lda]=attr;
            col++;
        }while(line!=NULL && word!=NULL && col<COLS);
        row++;        
    }  
}

int main(int argc, char *argv[])
{
    
    printf("with my modifications \n");
    hipsolverHandle_t cudenseH = NULL;

    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess; 
    hipError_t cudaStat2 = hipSuccess; 
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess; 

    /*used for timing purposes*/
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    int lwork = 0;
    int info_gpu = 0;

    // Matrix definitions
    const int rows = ROWS;
    const int cols = COLS;
    const int mat_A_size = rows*cols;
    const int mat_S_size = MIN(rows,cols);
    const int mat_U_size = rows*rows;
    const int mat_VT_size = cols*cols;
    size_t size_A = mat_A_size*sizeof(float);
    size_t size_S = mat_S_size*sizeof(float);
    size_t size_U = mat_U_size*sizeof(float);
    size_t size_VT = mat_VT_size*sizeof(float);

    float *h_A = (float*)malloc(size_A);
    float *h_S = (float*)malloc(size_S);
    float *h_U = (float*)malloc(size_U);
    float *h_VT = (float*)malloc(size_VT);
    float *d_work = NULL;
    float *rwork = NULL;

    fill(h_A, mat_A_size);
    //readMatrixFromFile(h_A, rows);


    printf("A\n");
    print_matrix(rows, cols, h_A, rows, "A");
    printf("\n\n\n");



    // Create data structures for device
    cusolver_status = hipsolverDnCreate(&cudenseH);
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    float* d_A = NULL;
    cudaStat1 = hipMalloc((void**)&d_A, size_A);
    assert(hipSuccess == cudaStat1);

    float* d_S = NULL;
    cudaStat2 = hipMalloc((void**)&d_S, size_S);
    assert(hipSuccess == cudaStat2);

    float* d_U = NULL;
    cudaStat3 = hipMalloc((void**)&d_U, size_U);
    assert(hipSuccess == cudaStat3);

    float* d_VT = NULL;
    cudaStat4 = hipMalloc((void**)&d_VT, size_VT);
    assert(hipSuccess == cudaStat4);

    int *devInfo = NULL; // info in gpu (device copy)
    cudaStat4 = hipMalloc((void**)&devInfo, sizeof(int));
    assert(hipSuccess == cudaStat4);

    /* copy data to device */
    cudaStat1 = hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);

    /*================SOLVE FOR QR======================== */
    int qr_Lwork = 0;
    cusolver_status = hipsolverDnSgeqrf_bufferSize(
            cudenseH,
            rows,
            cols,
            d_A,
            rows,
            &qr_Lwork);
    assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    float *qr_d_work = NULL;
    float *qr_d_tau = NULL;
    int *qr_devInfo = NULL;
    cudaStat1 = hipMalloc((void**)&qr_d_work, sizeof(float)*qr_Lwork);
    cudaStat2 = hipMalloc((void**)&qr_d_tau, sizeof(float)*cols);
    assert(cudaStat1 == hipSuccess);
    assert(cudaStat2 == hipSuccess);

    cusolver_status = hipsolverDnSgeqrf(
            cudenseH,
            rows,
            cols,
            d_A,
            rows,
            qr_d_tau,
            qr_d_work,
            qr_Lwork,
            qr_devInfo);

    /* Bring ``A'' back to device to get R */
    cudaStat1 = hipMemcpy(h_A, d_A, size_A, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(qr_devInfo == 0);

    printf("New A\n");
    print_matrix(rows, cols, h_A, rows, "A");
    printf("\n\n\n");

    float *h_R = NULL;
    float *d_R = NULL;
    const int mat_R_size = cols*cols;
    size_t size_R = mat_R_size*sizeof(float);
    h_R = (float*)malloc(size_R);

    /* Fill up R */
    for(int i = 0; i < rows; i++) {
        for(int j = 0; j < cols; j++) {
            if(i <= j)
                h_R[i + j*cols] = h_A[i + j*cols];
            else
                h_R[i + j*cols] = 0.0;
        }
    }


    printf("R\n");
    print_matrix(cols, cols, h_R, cols, "R");
    printf("\n\n\n");

    /*================SOLVE FOR SVD======================= */

    /* calculate the sizes needed for pre-allocated buffer Lwork  */
    cusolver_status = hipsolverDnSgesvd_bufferSize(cudenseH, rows, cols, &lwork );
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);
    printf("lwork/buffer size=%d\n",lwork);

    /* allocate memory for buffer */
    cudaStat1 = hipMalloc((void**)&d_work, sizeof(float)*lwork);
    assert(hipSuccess == cudaStat1);
    cudaStat2 = hipMalloc((void**)&rwork, sizeof(float)*lwork);
    assert(hipSuccess == cudaStat2);

    /* computer SVD */
    char jobu = 'A'; // We do not want/need U
    char jobvt = 'A'; // We want all the vectors of VT
    int lda = rows;
    int ldu = rows;
    int ldvt = cols;
    
    /* printf("with  allocating memory for rwork!\n");*/
    hipEventRecord(start, 0);


    int n_iterations = N_ITER;
    for(int i = 0; i < n_iterations; i++) {
        cusolver_status = hipsolverDnSgesvd (cudenseH, jobu, jobvt, rows, cols, d_A, lda, d_S, d_U, ldu, d_VT, ldvt, d_work, lwork, rwork, devInfo);
        cudaStat1 = hipDeviceSynchronize();
    }




    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    assert(hipSuccess == cudaStat1);
    printf("hipsolverDnSgesvd status :\t");
    switch(cusolver_status)
      {
        case HIPSOLVER_STATUS_SUCCESS:
          printf("success\n");
          break;
        case HIPSOLVER_STATUS_NOT_INITIALIZED :
          printf("Library cuSolver not initialized correctly\n");
          break;
        case HIPSOLVER_STATUS_INVALID_VALUE:
          printf("Invalid parameters passed\n");
          break;
        case HIPSOLVER_STATUS_INTERNAL_ERROR:
          printf("Internal operation failed\n");
          break;
        case HIPSOLVER_STATUS_EXECUTION_FAILED:
          printf("Execution failed\n");
          break;
      }


    /* ================END of SVD Computation======================= */

    /*  check if SVD is good or not  */
    cudaStat1 =hipMemcpy(&info_gpu,devInfo,sizeof(int),hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    printf("after SVD: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu); 
    
    ///*  copy the solutions back to the host */
    cudaStat1 = hipMemcpy(h_A, d_A, size_A, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(h_U, d_U, size_U, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(h_S, d_S, size_S, hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(h_U, d_U, size_VT, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1); 
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    /* 
    printf("U\n");
    print_matrix(rows, rows, h_U, rows, "A");
    printf("\n\n\n");

    printf("S\n");
    print_matrix(rows, cols, h_S, rows, "S");
    printf("\n\n\n");

    printf("VT\n");
    print_matrix(cols, cols, h_VT, cols, "VT");
    printf("\n\n\n");

     
    printf("A\n");
    print_matrix(rows, rows, h_A, rows, "A");
    printf("\n\n\n");
    */

    /* free resources */
    if (d_A ) hipFree(d_A);
    if (d_S ) hipFree(d_S);
    if (d_U ) hipFree(d_U);
    if (d_VT ) hipFree(d_VT);

    if(h_A) free(h_A);
    if(h_S) free(h_S);
    if(h_U) free(h_U);
    if(h_VT) free(h_VT);

    /*  print the time */
    hipEventElapsedTime(&time, start, stop);
    printf ("Time for the kernel: %f ms\n", time);
    printf ("\n\n\n");

    /*  total resources and computing */
    float Mflop_rate;
    Mflop_rate = 1e-6 * 4 * rows * rows * cols * n_iterations / time;
    printf ("n_iterations = %d\n",n_iterations);
    printf ("Mflop/s: %f\n", Mflop_rate);

    if (cudenseH) hipsolverDnDestroy(cudenseH);

    

    hipDeviceReset();
}
